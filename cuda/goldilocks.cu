#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <math.h>

// utilities and system includes
//#include <shrUtils.h>
#include "log.h"
// CUDA-C includes
#include <hip/hip_runtime_api.h>

// includes, project
//#include <cutil_inline.h> deprecated since CUDA 5.0
#include <hip/hip_runtime_api.h>

// includes, kernels
#include <goldilocks_kernel.cu>
#include <eraser_kernel.cu>
#include <atomicity_kernel.cu>


/************************************************/
#ifdef __cplusplus
extern "C" {
#endif
/************************************************/

#ifdef CHECK_AT_GPU

/********************************************/
// globals:

static inline void waitForKernel(hipEvent_t stop) {
	hipError_t err;
	while((err = hipEventQuery(stop)) != hipSuccess) {
		ASSERT (err == hipErrorNotReady);
		SLEEP(10);
	}
}

// timer
unsigned int timer = 0;

#if (MEMORY_MODEL == TEXTURE_MEMORY_MODEL)

	// host memory
	Event* h_block;
	
	// channel descriptor
	hipChannelFormatDesc channelDesc;
	
	// cuda array
	hipArray* cu_array;

#elif (MEMORY_MODEL == SHARED_MEMORY_MODEL) 
	// host memory
	Event* h_block;
	// device memory
	Event* d_block;
	
	hipStream_t streams[NUM_CUDA_STREAMS];
#endif

	
// memory for output
IndexPairList* h_indexPairs[NUM_CUDA_STREAMS];

IndexPairList* d_indexPairs[NUM_CUDA_STREAMS];

/********************************************/
/********************************************/

// setup the host and device memory
void initRaceChecker()
{
	checkCudaErrors( hipSetDeviceFlags(hipDeviceScheduleYield) );
	
	BloomFilter bloom_tmp;
	size_t sizeof_block = sizeof(Event) * BLOCK_SIZE;
	
	// init the timer
	//cutilCheckError( cutCreateTimer( &timer ) );
	
#if (MEMORY_MODEL == TEXTURE_MEMORY_MODEL) 
	
	size_t width = CHECKED_BLOCK_SIZE;
	size_t height = NUM_CONCURRENT_KERNELS;


	// init host memory
	h_block = NULL;
	checkCudaErrors( hipHostAlloc( (void**)&h_block, sizeof_block, hipHostMallocWriteCombined) );
	
    // allocate array and copy image data
	channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindSigned);
	checkCudaErrors( hipMallocArray( &cu_array, &channelDesc, width, height )); 
	
	// set texture parameters
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;    // access with normalized texture coordinates

	// Bind the array to the texture
	checkCudaErrors( hipBindTextureToArray(tex, cu_array, channelDesc));
	
#elif (MEMORY_MODEL == SHARED_MEMORY_MODEL)
	
	// init host memory
	h_block = NULL;
	checkCudaErrors( hipHostAlloc( (void**)&h_block, sizeof_block, hipHostMallocWriteCombined) );
	
	// init device memory
	checkCudaErrors(hipMalloc((void**) &d_block,  sizeof_block));
	
	if(NUM_CUDA_STREAMS > 1) {
		for (int i = 0; i < NUM_CUDA_STREAMS; ++i) {
			hipStreamCreate(&streams[i]);
		}
	} else {
		streams[0] = 0;
	}
	
#endif
		
	
	for(int i = 0; i < NUM_CUDA_STREAMS; ++i) {
		// setup the output memory for host
		checkCudaErrors(hipHostAlloc((void**) &h_indexPairs[i],  sizeof(IndexPairList), hipHostMallocWriteCombined));
	
		// setup the putput memory for device
		checkCudaErrors(hipMalloc((void**) &d_indexPairs[i],  sizeof(IndexPairList)));
	}
	
	// initialize the bloom filter 
	ASSERT(sizeof(BloomFilter) == sizeof(BloomKernelFilter));
	bloom_clear(&bloom_tmp);
	hipMemcpyToSymbol(HIP_SYMBOL("d_racyVars"), &bloom_tmp, sizeof(BloomKernelFilter), 0, hipMemcpyHostToDevice);
//	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL("d_racyVars"), &bloom_tmp, sizeof(BloomKernelFilter), 0, hipMemcpyHostToDevice));
}

/********************************************/
/********************************************/

// dealloc the host and device memory
void finalizeRaceChecker()
{
	hipEvent_t stop;
	checkCudaErrors( hipEventCreate( &stop ) );
	waitForKernel(stop);
	// remove the timer
	//cutilCheckError( cutDeleteTimer( timer));
	checkCudaErrors( hipEventDestroy(stop) );
	
#if (MEMORY_MODEL == TEXTURE_MEMORY_MODEL) 
	
	// Unbind the array from the texture
	checkCudaErrors( hipUnbindTexture(tex) );

	// free host memory
	checkCudaErrors( hipHostFree(h_block) );
	
	// free cuda memory
	checkCudaErrors( hipFreeArray(cu_array) );
	
#elif (MEMORY_MODEL == SHARED_MEMORY_MODEL)

	// free host memory
	checkCudaErrors( hipHostFree(h_block));
	// free device memory
	checkCudaErrors( hipFree(d_block) );
	if(NUM_CUDA_STREAMS > 1) {
		for (int i = 0; i < NUM_CUDA_STREAMS; ++i) {
			hipStreamDestroy(streams[i]);
		}
	}
#endif
	
	for(int i = 0; i < NUM_CUDA_STREAMS; ++i) {
		// free output memory for host
		checkCudaErrors( hipHostFree(h_indexPairs[i]) );
		
		// free output memory for device
		checkCudaErrors( hipFree(d_indexPairs[i]) );
	}
}

/********************************************/
/********************************************/

// Event == int4
IndexPairList** raceChecker(Block* block, size_t num_events)
{
	
//	size_t num_events = block->size;
//	if(num_events > BLOCK_SIZE) {
//		num_events = BLOCK_SIZE;
//	}
	size_t width = CHECKED_BLOCK_SIZE;
	size_t height = num_events / width;
	if(height <= 0) return NULL; //#!#
	//return NULL;
	
	unsigned int timer = 0;
    float elapsedTimeInMs = 0.0f;
    hipEvent_t start, stop;
    checkCudaErrors( hipEventCreate( &start ) );
    checkCudaErrors( hipEventCreate( &stop ) );
    
    //cutilCheckError( cutStartTimer(timer));
    checkCudaErrors( hipEventRecord(start, 0 ) );
    
    // reset d_indexPairs->size (important that size is the first field of IndexPairList
    for(int i = 0; i < NUM_CUDA_STREAMS; ++i) {
		unsigned int zero = 0; 
		checkCudaErrors(hipMemcpyAsync(&d_indexPairs[i]->size, &zero, sizeof(unsigned int), hipMemcpyHostToDevice, NULL));
    }
    
//-----------------------------------------------------------------
// prepare the memory
//-----------------------------------------------------------------
#if (MEMORY_MODEL == SHARED_MEMORY_MODEL)	
    size_t sizeof_block = num_events * sizeof(Event);
    ASSERT(sizeof_block > 0);
    
	//initialize the host memory
	memcpy((void*)h_block, (void*)block->events, sizeof_block);
	 
	// checkCudaErrors(hipMemcpyAsync((void*)d_block, (void*)h_block, sizeof_block, hipMemcpyHostToDevice, NULL));

	size_t sizeof_half_block = sizeof_block >> 1;
	ASSERT (sizeof_half_block > 0);
	
	for (int i = 0; i < NUM_CUDA_STREAMS; ++i) { 
		// copy to device
		Event* d_block_v = &d_block[(i * (num_events / NUM_CUDA_STREAMS))];  // ((void*)d_block) + (i * sizeof_half_block);
		Event* h_block_v = &h_block[(i * (num_events / NUM_CUDA_STREAMS))];  // ((void*)h_block) + (i * sizeof_half_block);
		
		checkCudaErrors(hipMemcpyAsync((void*)d_block_v, (void*)h_block_v, sizeof_half_block, hipMemcpyHostToDevice, streams[i]));
		
		//-----------------------------------------------------------------
		// call the kernel #!# after synchronization 
		//-----------------------------------------------------------------
		waitForKernel(start);
		checkCudaErrors( hipEventRecord(stop, 0 ) );
		
		if(glbConfig.algorithm == Goldilocks) {
			raceCheckerKernelGoldilocks <<< (height / NUM_CUDA_STREAMS), NUM_THREADS, 0, streams[i] >>> (d_block_v, CHECKED_BLOCK_SIZE, 0, d_indexPairs[i]);
			raceCheckerKernelGoldilocks <<< (height / NUM_CUDA_STREAMS), NUM_THREADS, 0, streams[i] >>> (d_block_v, CHECKED_BLOCK_SIZE, (CHECKED_BLOCK_SIZE >> 1), d_indexPairs[i]);
		} 
		else 
		if(glbConfig.algorithm == Eraser){
			raceCheckerKernelEraser <<< (height >> 1), NUM_THREADS, 0, streams[i] >>> (d_block_v, CHECKED_BLOCK_SIZE, 0, d_indexPairs[i]);
			raceCheckerKernelEraser <<< (height >> 1), NUM_THREADS, 0, streams[i] >>> (d_block_v, CHECKED_BLOCK_SIZE, (CHECKED_BLOCK_SIZE >> 1), d_indexPairs[i]);
		}
		
		// read the number of races
		checkCudaErrors(hipMemcpyAsync(&h_indexPairs[i]->size, &d_indexPairs[i]->size,  sizeof(unsigned int), hipMemcpyDeviceToHost, streams[i]));
		
		if(height < NUM_CUDA_STREAMS) break; // if there is only one checked block, then there is only one iteration
	}
    	
#else 
	#if (MEMORY_MODEL == TEXTURE_MEMORY_MODEL) 
	
		size_t sizeof_block = height * width * sizeof(Event);
		
		//initialize the memory
		memcpy((void*)h_block, (void*)block->events, sizeof_block);
			
		// copy image data
		checkCudaErrors( cudaMemcpyToArrayAsync( cu_array, 0, 0, (void*)h_block, sizeof_block, hipMemcpyHostToDevice, NULL));

	#elif (MEMORY_MODEL == CONSTANT_MEMORY_MODEL)
	
		size_t sizeof_block = num_events * sizeof(Event);
		
		hipMemcpyToSymbolAsync(HIP_SYMBOL("events"), block->events, num_events * sizeof(Event), 0, hipMemcpyHostToDevice, NULL);
//		checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL("events"), block->events, num_events * sizeof(Event), 0, hipMemcpyHostToDevice, NULL));
     
	#endif
		
//-----------------------------------------------------------------
// call the kernel
//-----------------------------------------------------------------
#if NUM_CUDA_STREAMS != 1
#error "NUM_CUDA_STREAMS must be 1 for texture and constant memory"
#endif
		
	if(glbConfig.algorithm == Goldilocks) {
		raceCheckerKernelGoldilocks <<< height, NUM_THREADS >>> (CHECKED_BLOCK_SIZE, 0, d_indexPairs[0]);
		raceCheckerKernelGoldilocks <<< height, NUM_THREADS >>> (CHECKED_BLOCK_SIZE, (CHECKED_BLOCK_SIZE >> 1) , d_indexPairs[0]);
	} 
	else 
	if(glbConfig.algorithm == Eraser){
		raceCheckerKernelEraser <<< height, NUM_THREADS >>> (CHECKED_BLOCK_SIZE, 0, d_indexPairs[0]);
		raceCheckerKernelEraser <<< height, NUM_THREADS >>> (CHECKED_BLOCK_SIZE, (CHECKED_BLOCK_SIZE >> 1) , d_indexPairs[0]);
	}
	
	// read the number of races
	checkCudaErrors(hipMemcpyAsync(&h_indexPairs[0]->size, &d_indexPairs[0]->size,  sizeof(unsigned int), hipMemcpyDeviceToHost, NULL));
	
#endif
   	///*
//-----------------------------------------------------------------
// get the results of the check
//-----------------------------------------------------------------
   	checkCudaErrors( hipEventRecord( stop, 0 ) );

	waitForKernel(stop); // wait for all streams
	
//-----------------------------------------------------------------
// check atomicity
//-----------------------------------------------------------------
#if ATOMICITY_ENABLED
	hipEvent_t stop2;
	checkCudaErrors( hipEventCreate( &stop2 ) );
#if (MEMORY_MODEL != ASYNC_SHARED_MEMORY_MODEL)	
	atomicityCheckerKernel <<< height, NUM_THREADS >>> (
		#if (MEMORY_MODEL == SHARED_MEMORY_MODEL)   
				d_block, 
		#endif 
				CHECKED_BLOCK_SIZE, d_indexPairs);
	
	checkCudaErrors( hipEventRecord( stop2, 0 ) );
	waitForKernel(stop2);
#else
	// code for async memory
#endif
	checkCudaErrors( hipEventDestroy(stop2) );
#endif // ATOMICITY_ENABLED
	
    //total elapsed time in ms
    //cutilCheckError( cutStopTimer( timer));
    //checkCudaErrors( hipEventElapsedTime( &elapsedTimeInMs, start, stop ) );
    //cutilCheckError( cutResetTimer( timer));

    int num_races = 0;
#if (MEMORY_MODEL == SHARED_MEMORY_MODEL) && (NUM_CUDA_STREAMS > 1)
    hipEvent_t stop3;
	checkCudaErrors( hipEventCreate( &stop3 ) );
    for (int i = 0; i < NUM_CUDA_STREAMS; ++i) {
		if(h_indexPairs[i]->size > 0) {
			num_races += h_indexPairs[i]->size; 
			IF_DEBUG(printf("%u races detected\n", h_indexPairs[i]->size));
			IF_DEBUG(fflush(stdout));
			checkCudaErrors(hipMemcpyAsync(&h_indexPairs[i]->pairs, &d_indexPairs[i]->pairs,  (h_indexPairs[i]->size * sizeof(IndexPair)), hipMemcpyDeviceToHost, streams[i]));
		
		}
    }
    checkCudaErrors( hipEventRecord( stop3, 0 ) );
	waitForKernel(stop3);
	checkCudaErrors( hipEventDestroy(stop3) );
#else
	if(h_indexPairs[0]->size > 0) {
		num_races += h_indexPairs[0]->size; 
		IF_DEBUG(printf("%u races detected\n", h_indexPairs[0]->size));
		IF_DEBUG(fflush(stdout));
		checkCudaErrors(hipMemcpy(&h_indexPairs[0]->pairs, &d_indexPairs[0]->pairs,  (h_indexPairs[0]->size * sizeof(IndexPair)), hipMemcpyDeviceToHost));
	
	}
#endif
    
    //clean up memory
	checkCudaErrors( hipEventDestroy(stop) );
	checkCudaErrors( hipEventDestroy(start) );
    
    return (num_races > 0 ? h_indexPairs : NULL);
//*/return NULL;
}

/*****************************************************/


int deviceQuery()
{
    shrSetLogFileName ("deviceQuery.txt");
    printf("Starting...\n\n");
    printf(" CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");

    int deviceCount = 0;
	if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
		printf("hipGetDeviceCount FAILED CUDA Driver and Runtime version may be mismatched.\n");
		printf("\nFAILED\n");
		return EXIT_FAILURE;// shrEXIT(argc, argv);
	}

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");

    int dev;
	int driverVersion = 0, runtimeVersion = 0;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        if (dev == 0) {
			// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
            if (deviceProp.major == 9999 && deviceProp.minor == 9999)
                printf("There is no device supporting CUDA.\n");
            else if (deviceCount == 1)
                printf("There is 1 device supporting CUDA\n");
            else
                printf("There are %d devices supporting CUDA\n", deviceCount);
        }
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

    #if CUDART_VERSION >= 2020
        // Console log
		hipDriverGetVersion(&driverVersion);
		printf("  CUDA Driver Version:                           %d.%d\n", driverVersion/1000, driverVersion%100);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("  CUDA Runtime Version:                          %d.%d\n", runtimeVersion/1000, runtimeVersion%100);
    #endif
        printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

		char msg[256];
		sprintf(msg, "  Total amount of global memory:                 %llu bytes\n", (unsigned long long) deviceProp.totalGlobalMem);
		printf("%s", msg);
    #if CUDART_VERSION >= 2000
        printf("  (%2d) Multiprocessors x (%2d) CUDA Cores/MP:     %d CUDA Cores\n",
			deviceProp.multiProcessorCount,
			ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
    #endif
        printf("  Total amount of constant memory:               %ld bytes\n", deviceProp.totalConstMem);
        printf("  Total amount of shared memory per block:       %ld bytes\n", deviceProp.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
        printf("  Warp size:                                     %d\n", deviceProp.warpSize);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("  Maximum memory pitch:                          %ld bytes\n", deviceProp.memPitch);
    #if CUDART_VERSION >= 4000
//		printf("  Memory Bus Width:                              %d-bit\n", deviceProp.memBusWidth);
//		printf("  Memory Clock rate:                             %.2f Mhz\n", deviceProp.memoryClock * 1e-3f);
    #endif

		printf("  Texture alignment:                             %ld bytes\n", deviceProp.textureAlignment);
        printf("  Clock rate:                                    %.2f GHz\n", deviceProp.clockRate * 1e-6f);
    #if CUDART_VERSION >= 2000
        printf("  Concurrent copy and execution:                 %s\n", deviceProp.deviceOverlap ? "Yes" : "No");
    #endif
    #if CUDART_VERSION >= 4000
		printf("  # of Asynchronous Copy Engines:                %d\n", deviceProp.asyncEngineCount);
    #endif
    #if CUDART_VERSION >= 2020
        printf("  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
        printf("  Integrated:                                    %s\n", deviceProp.integrated ? "Yes" : "No");
        printf("  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
        printf("  Compute mode:                                  %s\n", deviceProp.computeMode == hipComputeModeDefault ?
			                                                            "Default (multiple host threads can use this device simultaneously)" :
		                                                                deviceProp.computeMode == hipComputeModeExclusive ?
																		"Exclusive (only one host thread at a time can use this device)" :
		                                                                deviceProp.computeMode == hipComputeModeProhibited ?
																		"Prohibited (no host thread can use this device)" :
																		"Unknown");
    #endif
    #if CUDART_VERSION >= 3000
        printf("  Concurrent kernel execution:                   %s\n", deviceProp.concurrentKernels ? "Yes" : "No");
    #endif
    #if CUDART_VERSION >= 3010
        printf("  Device has ECC support enabled:                %s\n", deviceProp.ECCEnabled ? "Yes" : "No");
    #endif
    #if CUDART_VERSION >= 3020
		printf("  Device is using TCC driver mode:               %s\n", deviceProp.tccDriver ? "Yes" : "No");
    #endif

	}

    // csv masterlog info
    // *****************************
    // exe and CUDA driver name
    printf("\n");
	std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
    char cTemp[10];

    // driver version
    sProfileString += ", CUDA Driver Version = ";
    #ifdef WIN32
	    sprintf_s(cTemp, 10, "%d.%d", driverVersion/1000, driverVersion%100);
    #else
	    sprintf(cTemp, "%d.%d", driverVersion/1000, driverVersion%100);
    #endif
    sProfileString +=  cTemp;

    // Runtime version
    sProfileString += ", CUDA Runtime Version = ";
    #ifdef WIN32
	    sprintf_s(cTemp, 10, "%d.%d", runtimeVersion/1000, runtimeVersion%100);
    #else
	    sprintf(cTemp, "%d.%d", runtimeVersion/1000, runtimeVersion%100);
    #endif
    sProfileString +=  cTemp;

    // Device count
    sProfileString += ", NumDevs = ";
    #ifdef WIN32
        sprintf_s(cTemp, 10, "%d", deviceCount);
    #else
        sprintf(cTemp, "%d", deviceCount);
    #endif
    sProfileString += cTemp;

    // First 2 device names, if any
    for (dev = 0; dev < ((deviceCount > 2) ? 2 : deviceCount); ++dev)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        sProfileString += ", Device = ";
        sProfileString += deviceProp.name;
    }
    sProfileString += "\n";
    //printfEx(LOGBOTH | MASTER, 0, sProfileString.c_str());

    // finish
    printf("\n\nPASSED\n");
    return EXIT_SUCCESS; // shrEXIT(argc, argv);
}

#endif // CHECK_AT_GPU

/************************************************/
#ifdef __cplusplus
}
#endif
/************************************************/
